#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/Exceptions.h>
#include <ATen/cuda/HIPContext.h>

__global__ void update_scale_hysteresis_cuda_kernel(float* current_scale,
                                                    int* growth_tracker,
                                                    int* hysteresis_tracker,
                                                    const float* found_inf,
                                                    double growth_factor,
                                                    double backoff_factor,
                                                    int growth_interval,
                                                    int hysteresis)
{
  if (*found_inf > 0) {
    *hysteresis_tracker -= 1;

    // Only reset the growth tracker when hysteresis is larger than zero
    if (*hysteresis_tracker > 0) {
      *growth_tracker = 0;
      return;
    }
  }

  if (*found_inf) {
    *current_scale = (*current_scale)*backoff_factor;
    *growth_tracker = 0;
  } else {
    // Entering this branch means we just carried out a successful step,
    // so growth_tracker is incremented before comparing to growth_interval.
    auto successful = (*growth_tracker) + 1;
    if (successful == growth_interval) {
      auto new_scale = static_cast<float>((*current_scale)*growth_factor);
      // Do not grow the scale past fp32 bounds to inf.
      if (isfinite(new_scale)) {
          *current_scale = new_scale;
      }
      *growth_tracker = 0;
    } else {
      *growth_tracker = successful;
    }
  }

  // Reset the hysteresis tracker if no infs are found
  if (*found_inf <= 0) {
    *hysteresis_tracker = hysteresis;
  }
}

at::Tensor update_scale_hysteresis_cuda(at::Tensor current_scale,
                                        at::Tensor growth_tracker,
                                        at::Tensor hysteresis_tracker,
                                        at::Tensor found_inf,
                                        const double growth_factor,
                                        const double backoff_factor,
                                        const int64_t growth_interval,
                                        const int hysteresis)
{
  // 使用 data_ptr 来获取指针，并通过 const关键字控制只读性
  float* current_scale_ptr = current_scale.data_ptr<float>();
  int* growth_tracker_ptr = growth_tracker.data_ptr<int>();
  int* hysteresis_tracker_ptr = hysteresis_tracker.data_ptr<int>();
  const float* found_inf_ptr = found_inf.data_ptr<float>(); // 这里我们手动加上 const

  update_scale_hysteresis_cuda_kernel<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
    current_scale_ptr,
    growth_tracker_ptr,
    hysteresis_tracker_ptr,
    found_inf_ptr,
    growth_factor,
    backoff_factor,
    growth_interval,
    hysteresis);

  AT_CUDA_CHECK(hipGetLastError());

  return current_scale;
}

